#include "benchmark/benchmark.h"

#include "GaussianFilter.cuh"
#include "Image.cuh"
#include "Math.cuh"
#include "ImageApplyFilter.cuh"
#include "ThrustFunctors.cuh"

#include <thrust/device_vector.h>
const char *filename    = "../../images/rose_ml.jpg";

using namespace ced::gpu;
static void convertToGrayScale(benchmark::State& state)
{
    ced::gpu::Image img(filename);
    std::vector<float> m_red = img.getRedChannel();
    std::vector<float> m_green = img.getGreenChannel();
    std::vector<float> m_blue = img.getBlueChannel();
    // allocate to device
    thrust::device_vector<float> d_red = m_red;
    thrust::device_vector<float> d_green = m_green;
    thrust::device_vector<float> d_blue = m_blue;
    thrust::device_vector<float> d_result(m_red.size());
    for(auto _ : state)
    {
        auto start = std::chrono::high_resolution_clock::now();
        // sum red and green
        thrust::transform(  thrust::make_zip_iterator(thrust::make_tuple(d_red.begin(), d_green.begin(), d_blue.begin())),
                            thrust::make_zip_iterator(thrust::make_tuple(d_red.end(), d_green.end(), d_blue.end())),
                            d_result.begin(),
                            add_three_vectors()); 
        // DIVIDE
        thrust::transform(  d_result.begin(), 
                            d_result.end(), 
                            d_result.begin(), 
                            divideByConstant<float>(3.0f));
        // copy back to host
        auto end = std::chrono::high_resolution_clock::now();
        auto elapsed_seconds = std::chrono::duration_cast<std::chrono::duration<double>>(end-start);
        state.SetIterationTime(elapsed_seconds.count());

    }
}
BENCHMARK(convertToGrayScale)->UseManualTime();
//  ------------------------------------------------------------------------------------------------ 
static void createGaussianFilter(benchmark::State& state)
{
    float _sigma = 1.4f;
    int _dimension = 5;
    const float sigma2 = _sigma * _sigma;
    thrust::device_vector<float> filter(_dimension * _dimension);
    thrust::device_vector<int> itr(_dimension * _dimension);
    for(auto _ : state)
    {
        auto start = std::chrono::high_resolution_clock::now();
        thrust::sequence(itr.begin(), itr.end());
        thrust::transform(itr.begin(), itr.end(), filter.begin(), g(_dimension, sigma2)); 
        float sum = thrust::reduce(filter.begin(), filter.end());
        thrust::transform(filter.begin(), filter.end(), filter.begin(), multiplyConst(sum));
        auto end = std::chrono::high_resolution_clock::now();
        auto elapsed_seconds = std::chrono::duration_cast<std::chrono::duration<double>>(end-start);
        state.SetIterationTime(elapsed_seconds.count());

    }
}
BENCHMARK(createGaussianFilter)->UseManualTime();
//  ------------------------------------------------------------------------------------------------ 
static void applyFilter_Kernel(benchmark::State& state)
{
    ced::gpu::Image img(filename);
    int m_width = img.getWidth();
    int m_height = img.getHeight();
    std::vector<float> m_red = img.getRedChannel();
    std::vector<float> m_green = img.getGreenChannel();
    std::vector<float> m_blue = img.getBlueChannel();
    img.convertToGrayscale();
    std::vector<float> _filter = ced::gpu::gaussianFilter(5, 1.4f);
    int _dimension = 5;
    int nwidth = m_width - _dimension  + 1;
    int nheight = m_height - _dimension + 1;
    for(auto _ : state)
    {
        thrust::device_vector<float> d_ored     = m_red;
        thrust::device_vector<float> d_ogreen   = m_green;
        thrust::device_vector<float> d_oblue    = m_blue;

        thrust::device_vector<float> d_nred(nheight * nwidth);
        thrust::device_vector<float> d_ngreen(nheight * nwidth);
        thrust::device_vector<float> d_nblue(nheight * nwidth);

        thrust::device_vector<float> d_filter = _filter;            
        // --------------------typecast raw ptr-----------------------------------
        float* d_ored_ptr   = thrust::raw_pointer_cast(d_ored.data());
        float* d_ogreen_ptr = thrust::raw_pointer_cast(d_ogreen.data());
        float* d_oblue_ptr  = thrust::raw_pointer_cast(d_oblue.data());

        float* d_nred_ptr    = thrust::raw_pointer_cast(d_nred.data());
        float* d_ngreen_ptr  = thrust::raw_pointer_cast(d_ngreen.data());
        float* d_nblue_ptr   = thrust::raw_pointer_cast(d_nblue.data());

        float* d_filter_ptr = thrust::raw_pointer_cast(d_filter.data());
        // --------------------execution config-----------------------------------
        int blockW = 32;
        int blockH = 32;
        const dim3 grid(iDivUp(nwidth, blockW),
                        iDivUp(nheight, blockH));
        const dim3 threadBlock(blockW, blockH);


        auto start = std::chrono::high_resolution_clock::now();
        d_applyFilter<<<grid, threadBlock>>>(d_ored_ptr, 
                                             d_ogreen_ptr,
                                             d_oblue_ptr,
                                             d_nred_ptr,   
                                             d_ngreen_ptr,   
                                             d_nblue_ptr,   
                                             d_filter_ptr,
                                             nwidth,
                                             nheight, 
                                             _dimension);
        hipDeviceSynchronize();
        auto end = std::chrono::high_resolution_clock::now();
        auto elapsed_seconds = std::chrono::duration_cast<std::chrono::duration<double>>(end-start);
        state.SetIterationTime(elapsed_seconds.count());
    }
}
BENCHMARK(applyFilter_Kernel)->UseManualTime();
//  ------------------------------------------------------------------------------------------------ 
